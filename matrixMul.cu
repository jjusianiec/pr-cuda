#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__ void
MatrixMultiplyKernel_GlobalMem(float* C, const float* A, const float* B, unsigned int matrixDim)
{
	unsigned int squareBlockDim = blockDim.x;

	// Compute the row index
	unsigned int i = (squareBlockDim * blockIdx.y) + threadIdx.y;
	// Compute the column index
	unsigned int j = (squareBlockDim * blockIdx.x) + threadIdx.x;

	//unsigned int index = (i * matrixDim) + j;
	float sum = 0.0f;

	for (unsigned int x = i; x < matrixDim; x+=squareBlockDim)
	{
		for (unsigned int y = j; y < matrixDim; y+=squareBlockDim)
		{
			for (unsigned int k = 0; k < matrixDim; ++k)
			{
				sum += A[x * matrixDim + k] * B[k * matrixDim + y];
			}
			C[(x * matrixDim) + y] = sum;
			sum = 0;
		}
	}

	//for (unsigned int k = 0; k < matrixDim; ++k)
	//{
	//	sum += A[i * matrixDim + k] * B[k * matrixDim + j];
	//}
	//C[index] = sum;
}

template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = wA * BLOCK_SIZE * by;
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        __syncthreads();
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

template <int BLOCK_SIZE> __global__ void
matrixNeighboursMul(float *C, float *A, float *B, int wA, int wB)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = wA * BLOCK_SIZE * by;
	int aEnd = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	int bBegin = BLOCK_SIZE * bx;
	int bStep = BLOCK_SIZE * wB;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub[BLOCK_SIZE];
	for (int z = 0; z < BLOCK_SIZE; z++) {
		Csub[z] = 0;
	}

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin;
		a <= aEnd;
		a += aStep, b += bStep)
	{
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		__syncthreads();
#pragma unroll
		for (int z = 0; z < BLOCK_SIZE; ++z) {
			for (int k = 0; k < BLOCK_SIZE; ++k) {
				Csub[z] += As[ty][k] * Bs[k][z];
			}
		}

		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	for (int z = 0; z < BLOCK_SIZE; ++z) {
		int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
		C[c + wB * ty + z] = Csub[z];
	}
}

void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

int matrixMultiply(int argc, char **argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

#pragma region alloc host memory
	if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipError_t error;

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
#pragma endregion

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);
	//dim3 grid(1, 1);

    hipDeviceSynchronize();

#pragma region initEvents
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
#pragma endregion

    int nIter = 1;
    for (int j = 0; j < nIter; j++)
    {
		//MatrixMultiplyKernel_GlobalMem <<< grid, threads>>>(d_C, d_A, d_B, dimsA.x);
		//matrixMulCUDA<32><<<grid, threads>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
		matrixNeighboursMul<32> <<<grid, dim3(1, 32, 1)>>> (d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

#pragma region Error handling
    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
#pragma endregion

#pragma region countPerformance
	float msecPerMatrixMul = msecTotal / nIter;
	double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
		gigaFlops,
		msecPerMatrixMul,
		flopsPerMatrixMul,
		threads.x * threads.y);
#pragma endregion

#pragma region copy result to host
	// Copy result from device to host
	error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
#pragma endregion

#pragma region verifyResult

	/*for (int i = 0; i < dimsA.x; i++) {
		for (int j = 0; j < dimsA.x; j++) {
			std::cout << h_C[i + j] << " ";
		}
		std::cout << std::endl;
	}*/

	printf("Checking computed result for correctness: ");
	bool correct = true;

	// test relative error by the formula
	//     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
	double eps = 1.e-6; // machine zero

	for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
	{
		double abs_err = fabs(h_C[i] - (dimsA.x * valB));
		double dot_length = dimsA.x;
		double abs_val = fabs(h_C[i]);
		double rel_err = abs_err / abs_val / dot_length;

		if (rel_err > eps)
		{
			printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
			correct = false;
		}
	}
#pragma endregion

#pragma region clean
	printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	if (correct)
	{
		return EXIT_SUCCESS;
	}
	else
	{
		return EXIT_FAILURE;
	}
#pragma endregion
}

int main(int argc, char **argv)
{
#pragma region Init
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // Use a larger block size for Fermi and above
    int block_size = (deviceProp.major < 2) ? 16 : 32;
	std::cout << "Block size: " << block_size << "\n";
#pragma endregion

	int matrixDim = 10 * block_size;
    dim3 dimsA(matrixDim, matrixDim, 1);
    dim3 dimsB(matrixDim, matrixDim, 1);
	
#pragma region ReadArgs
    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA"))
    {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA"))
    {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB"))
    {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB"))
    {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);
#pragma endregion

    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);
	//system("pause");
}
